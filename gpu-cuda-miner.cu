#include <cstdint>
#include <hip/hip_runtime.h>

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

#if defined(_MSC_VER)
#define ALIGN(x) __declspec(align(x))
#else
#define ALIGN(x) __attribute__((aligned(x)))
#endif

#if __CUDA_ARCH__ >= 320
__device__ __forceinline__
uint64_t rotr64(const uint64_t value, const int offset)
{
	uint2 result;
	if(offset < 32)
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
	}
	else
	{
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(__double2hiint(__longlong_as_double(value))), "r"(__double2loint(__longlong_as_double(value))), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(__double2loint(__longlong_as_double(value))), "r"(__double2hiint(__longlong_as_double(value))), "r"(offset));
	}
	return __double_as_longlong(__hiloint2double(result.y, result.x));
}
#else
__device__ __forceinline__
uint64_t rotr64(const uint64_t x, const int offset)
{
	uint64_t result;
	asm("{\n\t"
		".reg .b64 lhs;\n\t"
		".reg .u32 roff;\n\t"
		"shr.b64 lhs, %1, %2;\n\t"
		"sub.u32 roff, 64, %2;\n\t"
		"shl.b64 %0, %1, roff;\n\t"
		"add.u64 %0, %0, lhs;\n\t"
		"}\n"
		: "=l"(result) : "l"(x), "r"(offset));
	return result;
}
#endif

#define blocksize 256
#define npt 64

__global__ void __launch_bounds__(blocksize, 4) nonceGrind(const uint64_t *const __restrict__ headerIn, uint64_t *const __restrict__ hashOut, uint64_t *const __restrict__ nonceOut, const uint64_t *const __restrict__ v1)
{
	uint64_t header[10], h[4], v[16];
	int i;

	const uint32_t id = (blockDim.x * blockIdx.x + threadIdx.x)*npt;

#pragma unroll
	for(i = 0; i < 10; i++)
		header[i] = headerIn[i];

	for(i = 0; i < npt; i++)
	{
		((uint32_t*)header)[8] = id + i;
		v[2] = 0x3c6ef372fe94f82bu + 0x1f83d9abfb41bd6bu + header[4]; v[14] = rotr64(~0x1f83d9abfb41bd6bu ^ v[2], 32); v[10] = 0x3c6ef372fe94f82bu + v[14]; v[6] = rotr64(0x1f83d9abfb41bd6bu ^ v[10], 24);
		v[2] = v[2] + v[6] + header[5]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = 0xa54ff53a5f1d36f1u + 0x5be0cd19137e2179u + header[6]; v[15] = rotr64(0x5be0cd19137e2179u ^ v[3], 32); v[11] = 0xa54ff53a5f1d36f1u + v[15]; v[7] = rotr64(0x5be0cd19137e2179u ^ v[11], 24);
		v[3] = v[3] + v[7] + header[7]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v1[0] + v1[5] + header[8]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v1[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[9]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v1[1] + v[6];            v[12] = rotr64(v1[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64(v1[13] ^ v[2], 32); v[8] = v1[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v1[4];            v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v1[9] + v[14]; v[4] = rotr64(v1[4] ^ v[9], 24);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[8]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[6]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[0]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[2]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[7]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[5]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[3]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[8]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[0]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[5]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[2]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[3]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[6]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[7]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[1]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[9]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[4]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[7]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[9]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[3]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[1]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[2]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[6]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[5]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[4]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[0]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[8]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[9]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[0]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[5]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[7]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[2]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[4]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[1]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[6]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[8]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[3]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[2]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[6]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];             v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[0]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[8]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[3]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[4]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[7]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[5]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[1]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[9]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[5]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[1]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];             v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[4]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[0]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[7]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[6]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[3]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[9]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[2]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[8]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];            v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[7]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5];             v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[1]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[3]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[9]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[5]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[0]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[4]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[8]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[6]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[2]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[6]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[9]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[3]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[0]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[8]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[2]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[7]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[1]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[4]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[5]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[2]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[8]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[4]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[7]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[6]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[1]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[5]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[9]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[3]; v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[0]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[0]; v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4] + header[1]; v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[2]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[3]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[4]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6] + header[5]; v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[6]; v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[7]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[8]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5] + header[9]; v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6];             v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4];             v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 32); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 24);
		v[0] = v[0] + v[4];             v[12] = rotr64(v[12] ^ v[0], 16); v[8] = v[8] + v[12]; v[4] = rotr64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[4]; v[13] = rotr64(v[13] ^ v[1], 32); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 24);
		v[1] = v[1] + v[5] + header[8]; v[13] = rotr64(v[13] ^ v[1], 16); v[9] = v[9] + v[13]; v[5] = rotr64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = rotr64(v[14] ^ v[2], 32); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 24);
		v[2] = v[2] + v[6];             v[14] = rotr64(v[14] ^ v[2], 16); v[10] = v[10] + v[14]; v[6] = rotr64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = rotr64(v[15] ^ v[3], 32); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 24);
		v[3] = v[3] + v[7] + header[6]; v[15] = rotr64(v[15] ^ v[3], 16); v[11] = v[11] + v[15]; v[7] = rotr64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1]; v[15] = rotr64(v[15] ^ v[0], 32); v[10] = v[10] + v[15]; v[5] = rotr64(v[5] ^ v[10], 24);
		v[0] = v[0] + v[5];             v[15] = rotr64(v[15] ^ v[0], 16); v[10] = v[10] + v[15];
		v[1] = v[1] + v[6] + header[0]; v[12] = rotr64(v[12] ^ v[1], 32); v[11] = v[11] + v[12]; v[6] = rotr64(v[6] ^ v[11], 24);
		v[1] = v[1] + v[6] + header[2]; v[12] = rotr64(v[12] ^ v[1], 16); v[11] = v[11] + v[12];
		v[2] = v[2] + v[7];             v[13] = rotr64(v[13] ^ v[2], 32); v[8] = v[8] + v[13]; v[7] = rotr64(v[7] ^ v[8], 24);
		v[2] = v[2] + v[7] + header[7]; v[13] = rotr64(v[13] ^ v[2], 16); v[8] = v[8] + v[13];
		v[3] = v[3] + v[4] + header[5]; v[14] = rotr64(v[14] ^ v[3], 32); v[9] = v[9] + v[14]; v[4] = rotr64(v[4] ^ v[9], 24);
		v[3] = v[3] + v[4] + header[3]; v[14] = rotr64(v[14] ^ v[3], 16); v[9] = v[9] + v[14];

		h[0] = 0x6A09E667F2BDC928 ^ v[0] ^ v[8];
		h[1] = 0xbb67ae8584caa73b ^ v[1] ^ v[9];
		h[2] = 0x3c6ef372fe94f82b ^ v[2] ^ v[10];
		h[3] = 0xa54ff53a5f1d36f1 ^ v[3] ^ v[11];
		if(*((uint32_t*)h) == 0)
		{
			*nonceOut = header[4];
#pragma unroll
			for(i = 0; i < 4; i++)
				hashOut[i] = h[i];
			return;
		}
	}
}

void nonceGrindcuda(hipStream_t cudastream, uint32_t threads, uint64_t *blockHeader, uint64_t *headerHash, uint64_t *nonceOut, uint64_t *vpre)
{
	nonceGrind <<<threads / blocksize / npt, blocksize, 0, cudastream >>>(blockHeader, headerHash, nonceOut, vpre);
}


